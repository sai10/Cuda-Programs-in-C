#include "hip/hip_runtime.h"
__global__`
void rgba_to_greyscale(const uchar4* const rgbaImage,
               unsigned char* const greyImage,
               int numRows, int numCols)
{`

int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;
if(pos_x >= numCols || pos_y >= numRows)
    return;

uchar4 rgba = rgbaImage[pos_x + pos_y * numCols];
greyImage[pos_x + pos_y * numCols] = (.299f * rgba.x + .587f * rgba.y + .114f * rgba.z); 

}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage,
                            uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage,
                            size_t numRows,
                            size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(numCols/32, numCols/32 , 1);  //TODO
  const dim3 gridSize(numRows/12, numRows/12 , 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage,
                                             d_greyImage,
                                             numRows,
                                             numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
