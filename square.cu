#include<stdio.h>
#include<hip/hip_runtime.h>

//	KERNEL
__global__ 
void Square(float * d_out, float * d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f*f;
}

int main(int argc, char ** argv){

	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

//	GENERATING INPUT ARRAY IN HOST
	float h_in[ARRAY_SIZE];
	for(int i=0;i<64;i++)
		h_in[i] = float(i);

	float h_out[ARRAY_SIZE];

//	DESCRIBING GPU MEMORY POINTERS
	float *d_in;
	float *d_out;

//	 ALLOCATING MEMORY IN GPU
	hipMalloc((void**)&d_in,ARRAY_BYTES);
	hipMalloc((void**)&d_out,ARRAY_BYTES);	

//	TRANSFER THE ARRAY TO GPU
	hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);
	
//	LAUNCH THE KERNEL
	Square<<<1,ARRAY_SIZE>>>(d_out,d_in);

//	TRANSFER THE ARRAY BACK TO CPU
	hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);

//	DISPLAYING THE RESULT
	for(int i=0; i<ARRAY_SIZE;i++){
		printf("%f",h_out[i]);
		printf(((i%4)!=3)?"\t":"\n");
	}
}
